#include "hip/hip_runtime.h"
#include "yak/kfusion/cuda/device.hpp"
#include "yak/kfusion/cuda/texture_binder.hpp"
#include "yak/mc/marching_cubes_tables.h"
//#include <stdio.h>

#include <thrust/device_vector.h>
#include <thrust/scan.h>

using namespace kfusion::device;

////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Volume initialization

namespace kfusion
{
    namespace device
    {
        __global__ void clear_volume_kernel(TsdfVolume tsdf)
        {
            int x = threadIdx.x + blockIdx.x * blockDim.x;
            int y = threadIdx.y + blockIdx.y * blockDim.y;

            if (x < tsdf.dims.x && y < tsdf.dims.y)
            {
                TsdfVolume::elem_type *beg = tsdf.beg(x, y);
                TsdfVolume::elem_type *end = beg + tsdf.dims.x * tsdf.dims.y * tsdf.dims.z;

                for (TsdfVolume::elem_type* pos = beg; pos != end; pos = tsdf.zstep(pos))
                    *pos = pack_tsdf(0.f, 0);
            }
        }
    }
}

void kfusion::device::clear_volume(TsdfVolume volume)
{
    dim3 block(32, 8);
    dim3 grid(1, 1, 1);
    grid.x = divUp(volume.dims.x, block.x);
    grid.y = divUp(volume.dims.y, block.y);

    clear_volume_kernel<<<grid, block>>>(volume);
    cudaSafeCall(hipGetLastError());
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Volume integration

namespace kfusion
{
    namespace device
    {
        texture<float, 2> dists_tex(0, hipFilterModePoint, hipAddressModeBorder, cudaCreateChannelDescHalf());

        struct TsdfIntegrator
        {
                Aff3f vol2cam;
                Projector proj;
                int2 dists_size;

                float tranc_dist_inv;

                __kf_device__
                void operator()(TsdfVolume& volume) const
                {
                    int x = blockIdx.x * blockDim.x + threadIdx.x;
                    int y = blockIdx.y * blockDim.y + threadIdx.y;

                    if (x >= volume.dims.x || y >= volume.dims.y)
                        return;

                    //float3 zstep = vol2cam.R * make_float3(0.f, 0.f, volume.voxel_size.z);
                    float3 zstep = make_float3(vol2cam.R.data[0].z, vol2cam.R.data[1].z, vol2cam.R.data[2].z) * volume.voxel_size.z;

                    float3 vx = make_float3(x * volume.voxel_size.x, y * volume.voxel_size.y, 0.0f);
                    float3 vc = vol2cam * vx; //tranform from volume coo frame to camera one


                    TsdfVolume::elem_type* vptr = volume.beg(x, y);
                    for (int i = 0; i < volume.dims.z; ++i, vc += zstep, vptr = volume.zstep(vptr))
                    {
                        float2 coo = proj(vc);

                        //#if defined __CUDA_ARCH__ && __CUDA_ARCH__ >= 300
                        // this is actually workaround for kepler. it doesn't return 0.f for texture
                        // fetches for out-of-border coordinates even for cudaaddressmodeborder mode
                        if (coo.x < 0 || coo.y < 0 || coo.x >= dists_size.x || coo.y >= dists_size.y)
                            continue;
                        //#endif

                        float Dp = tex2D(dists_tex, coo.x, coo.y);

                        if (Dp == 0 || vc.z <= 0)
                            continue;
                                        
                        float normv = __fsqrt_rn(dot(vc, vc));
                        float sdf = Dp - normv; 

                        if (sdf >= -volume.trunc_dist)
                        {
                            float tsdf = fmin(1.f, sdf * tranc_dist_inv);

                            //read and unpack
                            int weight_prev;
                            float tsdf_prev = unpack_tsdf(gmem::LdCs(vptr), weight_prev);

                            float tsdf_new = __fdividef(__fmaf_rn(tsdf_prev, weight_prev, tsdf), weight_prev + 1);
                            int weight_new = min(weight_prev + 1, volume.max_weight);

                            //pack and write
                            gmem::StCs(pack_tsdf(tsdf_new, weight_new), vptr);
                        }
                    }
                }
        };

        __global__ void integrate_kernel(const TsdfIntegrator integrator, TsdfVolume volume)
        {
            integrator(volume);
        }
    }
}

void kfusion::device::integrate(const Dists& dists, TsdfVolume& volume, const Aff3f& aff, const Projector& proj)
{
    TsdfIntegrator ti;
    ti.dists_size = make_int2(dists.cols, dists.rows);
    ti.vol2cam = aff;
    ti.proj = proj;
    ti.tranc_dist_inv = 1.f / volume.trunc_dist;

    dists_tex.filterMode = hipFilterModePoint;
    dists_tex.addressMode[0] = hipAddressModeBorder;
    dists_tex.addressMode[1] = hipAddressModeBorder;
    dists_tex.addressMode[2] = hipAddressModeBorder;
    TextureBinder binder(dists, dists_tex, cudaCreateChannelDescHalf());

    dim3 block(32, 8);
    dim3 grid(divUp(volume.dims.x, block.x), divUp(volume.dims.y, block.y));

    integrate_kernel<<<grid, block>>>(ti, volume);
    cudaSafeCall(hipGetLastError());
    cudaSafeCall(hipDeviceSynchronize());
}

////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/// Volume ray casting

namespace kfusion
{
    namespace device
    {
        __kf_device__ void intersect(float3 ray_org, float3 ray_dir, /*float3 box_min,*/float3 box_max, float &tnear, float &tfar)
        {
            const float3 box_min = make_float3(0.f, 0.f, 0.f);

            // compute intersection of ray with all six bbox planes
            float3 invR = make_float3(1.f / ray_dir.x, 1.f / ray_dir.y, 1.f / ray_dir.z);
            float3 tbot = invR * (box_min - ray_org);
            float3 ttop = invR * (box_max - ray_org);

            // re-order intersections to find smallest and largest on each axis
            float3 tmin = make_float3(fminf(ttop.x, tbot.x), fminf(ttop.y, tbot.y), fminf(ttop.z, tbot.z));
            float3 tmax = make_float3(fmaxf(ttop.x, tbot.x), fmaxf(ttop.y, tbot.y), fmaxf(ttop.z, tbot.z));

            // find the largest tmin and the smallest tmax
            tnear = fmaxf(fmaxf(tmin.x, tmin.y), fmaxf(tmin.x, tmin.z));
            tfar = fminf(fminf(tmax.x, tmax.y), fminf(tmax.x, tmax.z));
        }

        template<typename Vol>
        __kf_device__ float interpolate(const Vol& volume, const float3& p_voxels)
        {
            float3 cf = p_voxels;

            //rounding to negative infinity
            int3 g = make_int3(__float2int_rd(cf.x), __float2int_rd(cf.y), __float2int_rd(cf.z));

            if (g.x < 0 || g.x >= volume.dims.x - 1 || g.y < 0 || g.y >= volume.dims.y - 1 || g.z < 0 || g.z >= volume.dims.z - 1)
                return numeric_limits<float>::quiet_NaN();

            float a = cf.x - g.x;
            float b = cf.y - g.y;
            float c = cf.z - g.z;

            float tsdf = 0.f;
            tsdf += unpack_tsdf(*volume(g.x + 0, g.y + 0, g.z + 0)) * (1 - a) * (1 - b) * (1 - c);
            tsdf += unpack_tsdf(*volume(g.x + 0, g.y + 0, g.z + 1)) * (1 - a) * (1 - b) * c;
            tsdf += unpack_tsdf(*volume(g.x + 0, g.y + 1, g.z + 0)) * (1 - a) * b * (1 - c);
            tsdf += unpack_tsdf(*volume(g.x + 0, g.y + 1, g.z + 1)) * (1 - a) * b * c;
            tsdf += unpack_tsdf(*volume(g.x + 1, g.y + 0, g.z + 0)) * a * (1 - b) * (1 - c);
            tsdf += unpack_tsdf(*volume(g.x + 1, g.y + 0, g.z + 1)) * a * (1 - b) * c;
            tsdf += unpack_tsdf(*volume(g.x + 1, g.y + 1, g.z + 0)) * a * b * (1 - c);
            tsdf += unpack_tsdf(*volume(g.x + 1, g.y + 1, g.z + 1)) * a * b * c;
            return tsdf;
        }

        struct TsdfRaycaster
        {
                TsdfVolume volume;

                Aff3f aff;
                Mat3f Rinv;

                Vec3f volume_size;
                Reprojector reproj;
                float time_step;
                float3 gradient_delta;
                float3 voxel_size_inv;

                TsdfRaycaster(const TsdfVolume& volume, const Aff3f& aff, const Mat3f& Rinv, const Reprojector& _reproj);

                __kf_device__
                float fetch_tsdf(const float3& p) const
                {
                    //rounding to nearest even
                    int x = __float2int_rn(p.x * voxel_size_inv.x);
                    int y = __float2int_rn(p.y * voxel_size_inv.y);
                    int z = __float2int_rn(p.z * voxel_size_inv.z);
                    return unpack_tsdf(*volume(x, y, z));
                }

                __kf_device__
                void operator()(PtrStepSz<ushort> depth, PtrStep<Normal> normals) const
                {
                    int x = blockIdx.x * blockDim.x + threadIdx.x;
                    int y = blockIdx.y * blockDim.y + threadIdx.y;

                    if (x >= depth.cols || y >= depth.rows)
                        return;

                    const float qnan = numeric_limits<float>::quiet_NaN();

                    depth(y, x) = 0;
                    normals(y, x) = make_float4(qnan, qnan, qnan, qnan);

                    float3 ray_org = aff.t;
                    float3 ray_dir = normalized(aff.R * reproj(x, y, 1.f));

                    // We do subtract voxel size to minimize checks after
                    // Note: origin of volume coordinate is placeed
                    // in the center of voxel (0,0,0), not in the corener of the voxel!
                    float3 box_max = volume_size - volume.voxel_size;

                    float tmin, tmax;
                    intersect(ray_org, ray_dir, box_max, tmin, tmax);

                    const float min_dist = 0.f;
                    tmin = fmax(min_dist, tmin);
                    if (tmin >= tmax)
                        return;

                    tmax -= time_step;
                    float3 vstep = ray_dir * time_step;
                    float3 next = ray_org + ray_dir * tmin;

                    float tsdf_next = fetch_tsdf(next);
                    for (float tcurr = tmin; tcurr < tmax; tcurr += time_step)
                    {
                        float tsdf_curr = tsdf_next;
                        float3 curr = next;
                        next += vstep;

                        tsdf_next = fetch_tsdf(next);
                        if (tsdf_curr < 0.f && tsdf_next > 0.f)
                            break;

                        if (tsdf_curr > 0.f && tsdf_next < 0.f)
                        {
                            float Ft = interpolate(volume, curr * voxel_size_inv);
                            float Ftdt = interpolate(volume, next * voxel_size_inv);

                            float Ts = tcurr - __fdividef(time_step * Ft, Ftdt - Ft);

                            float3 vertex = ray_org + ray_dir * Ts;
                            float3 normal = compute_normal(vertex);

                            if (!isnan(normal.x * normal.y * normal.z))
                            {
                                normal = Rinv * normal;
                                vertex = Rinv * (vertex - aff.t);

                                normals(y, x) = make_float4(normal.x, normal.y, normal.z, 0);
                                depth(y, x) = static_cast<ushort>(vertex.z * 1000);
                            }
                            break;
                        }
                    } /* for (;;) */
                }

                __kf_device__
                void operator()(PtrStepSz<Point> points, PtrStep<Normal> normals) const
                {
                    int x = blockIdx.x * blockDim.x + threadIdx.x;
                    int y = blockIdx.y * blockDim.y + threadIdx.y;

                    if (x >= points.cols || y >= points.rows)
                        return;

                    const float qnan = numeric_limits<float>::quiet_NaN();

                    points(y, x) = normals(y, x) = make_float4(qnan, qnan, qnan, qnan);

                    float3 ray_org = aff.t;
                    float3 ray_dir = normalized(aff.R * reproj(x, y, 1.f));

                    // We do subtract voxel size to minimize checks after
                    // Note: origin of volume coordinate is placeed
                    // in the center of voxel (0,0,0), not in the corener of the voxel!
                    float3 box_max = volume_size - volume.voxel_size;

                    float tmin, tmax;
                    intersect(ray_org, ray_dir, box_max, tmin, tmax);

                    const float min_dist = 0.f;
                    tmin = fmax(min_dist, tmin);
                    if (tmin >= tmax)
                        return;

                    tmax -= time_step;
                    float3 vstep = ray_dir * time_step;
                    float3 next = ray_org + ray_dir * tmin;

                    float tsdf_next = fetch_tsdf(next);
                    for (float tcurr = tmin; tcurr < tmax; tcurr += time_step)
                    {
                        float tsdf_curr = tsdf_next;
                        float3 curr = next;
                        next += vstep;

                        tsdf_next = fetch_tsdf(next);
                        if (tsdf_curr < 0.f && tsdf_next > 0.f)
                            break;

                        if (tsdf_curr > 0.f && tsdf_next < 0.f)
                        {
                            float Ft = interpolate(volume, curr * voxel_size_inv);
                            float Ftdt = interpolate(volume, next * voxel_size_inv);

                            float Ts = tcurr - __fdividef(time_step * Ft, Ftdt - Ft);

                            float3 vertex = ray_org + ray_dir * Ts;
                            float3 normal = compute_normal(vertex);

                            if (!isnan(normal.x * normal.y * normal.z))
                            {
                                normal = Rinv * normal;
                                vertex = Rinv * (vertex - aff.t);

                                normals(y, x) = make_float4(normal.x, normal.y, normal.z, 0.f);
                                points(y, x) = make_float4(vertex.x, vertex.y, vertex.z, 0.f);
                            }
                            break;
                        }
                    } /* for (;;) */
                }

                __kf_device__
                float3 compute_normal(const float3& p) const
                {
                    float3 n;

                    float Fx1 = interpolate(volume, make_float3(p.x + gradient_delta.x, p.y, p.z) * voxel_size_inv);
                    float Fx2 = interpolate(volume, make_float3(p.x - gradient_delta.x, p.y, p.z) * voxel_size_inv);
                    n.x = __fdividef(Fx1 - Fx2, gradient_delta.x);

                    float Fy1 = interpolate(volume, make_float3(p.x, p.y + gradient_delta.y, p.z) * voxel_size_inv);
                    float Fy2 = interpolate(volume, make_float3(p.x, p.y - gradient_delta.y, p.z) * voxel_size_inv);
                    n.y = __fdividef(Fy1 - Fy2, gradient_delta.y);

                    float Fz1 = interpolate(volume, make_float3(p.x, p.y, p.z + gradient_delta.z) * voxel_size_inv);
                    float Fz2 = interpolate(volume, make_float3(p.x, p.y, p.z - gradient_delta.z) * voxel_size_inv);
                    n.z = __fdividef(Fz1 - Fz2, gradient_delta.z);

                    return normalized(n);
                }
        };

        inline TsdfRaycaster::TsdfRaycaster(const TsdfVolume& _volume, const Aff3f& _aff, const Mat3f& _Rinv, const Reprojector& _reproj) :
                volume(_volume), aff(_aff), Rinv(_Rinv), reproj(_reproj)
        {
        }

        __global__ void raycast_kernel(const TsdfRaycaster raycaster, PtrStepSz<ushort> depth, PtrStep<Normal> normals)
        {
            raycaster(depth, normals);
        }
        ;

        __global__ void raycast_kernel(const TsdfRaycaster raycaster, PtrStepSz<Point> points, PtrStep<Normal> normals)
        {
            raycaster(points, normals);
        }
        ;

    }
}

void kfusion::device::raycast(const TsdfVolume& volume, const Aff3f& aff, const Mat3f& Rinv, const Reprojector& reproj, Depth& depth, Normals& normals, float raycaster_step_factor, float gradient_delta_factor)
{
    TsdfRaycaster rc(volume, aff, Rinv, reproj);

    rc.volume_size = volume.voxel_size * volume.dims;
    rc.time_step = volume.trunc_dist * raycaster_step_factor;
    rc.gradient_delta = volume.voxel_size * gradient_delta_factor;
    rc.voxel_size_inv = 1.f / volume.voxel_size;

    dim3 block(32, 8);
    dim3 grid(divUp(depth.cols(), block.x), divUp(depth.rows(), block.y));

    raycast_kernel<<<grid, block>>>(rc, (PtrStepSz<ushort> ) depth, normals);
    cudaSafeCall(hipGetLastError());
}

void kfusion::device::raycast(const TsdfVolume& volume, const Aff3f& aff, const Mat3f& Rinv, const Reprojector& reproj, Points& points, Normals& normals, float raycaster_step_factor, float gradient_delta_factor)
{
    TsdfRaycaster rc(volume, aff, Rinv, reproj);

    rc.volume_size = volume.voxel_size * volume.dims;
    rc.time_step = volume.trunc_dist * raycaster_step_factor;
    rc.gradient_delta = volume.voxel_size * gradient_delta_factor;
    rc.voxel_size_inv = 1.f / volume.voxel_size;

    dim3 block(32, 8);
    dim3 grid(divUp(points.cols(), block.x), divUp(points.rows(), block.y));

    raycast_kernel<<<grid, block>>>(rc, (PtrStepSz<Point> ) points, normals);
    cudaSafeCall(hipGetLastError());
}

////////////////////////////////////////////////////////////////////////////////////////
/// Volume cloud exctraction

namespace kfusion
{
    namespace device
    {
        ////////////////////////////////////////////////////////////////////////////////////////
        ///// Prefix Scan utility

        enum ScanKind
        {
            exclusive, inclusive
        };

        template<ScanKind Kind, class T>
        __kf_device__ T scan_warp(volatile T *ptr, const unsigned int idx = threadIdx.x)
        {
            const unsigned int lane = idx & 31;       // index of thread in warp (0..31)

            if (lane >= 1)
                ptr[idx] = ptr[idx - 1] + ptr[idx];
            if (lane >= 2)
                ptr[idx] = ptr[idx - 2] + ptr[idx];
            if (lane >= 4)
                ptr[idx] = ptr[idx - 4] + ptr[idx];
            if (lane >= 8)
                ptr[idx] = ptr[idx - 8] + ptr[idx];
            if (lane >= 16)
                ptr[idx] = ptr[idx - 16] + ptr[idx];

            if (Kind == inclusive)
                return ptr[idx];
            else
                return (lane > 0) ? ptr[idx - 1] : 0;
        }

        __device__ int global_count = 0;
        __device__ int output_count;
        __device__ unsigned int blocks_done = 0;

        struct FullScan6
        {
                enum
                {
                    CTA_SIZE_X = 32, CTA_SIZE_Y = 6, CTA_SIZE = CTA_SIZE_X * CTA_SIZE_Y,

                    MAX_LOCAL_POINTS = 3
                };

                TsdfVolume volume;
                Aff3f aff;

                FullScan6(const TsdfVolume& vol) :
                        volume(vol)
                {
                }

                __kf_device__
                float fetch(int x, int y, int z, int& weight) const
                {
                    return unpack_tsdf(*volume(x, y, z), weight);
                }

                __kf_device__
                void operator ()(PtrSz<Point> output) const
                {
                    int x = threadIdx.x + blockIdx.x * CTA_SIZE_X;
                    int y = threadIdx.y + blockIdx.y * CTA_SIZE_Y;
#if __CUDA_ARCH__ < 200
                    __shared__ int cta_buffer[CTA_SIZE];
#endif

#if __CUDA_ARCH__ >= 120
                    if (__all_sync(0xFFFFFFFF, x >= volume.dims.x) || __all_sync(0xFFFFFFFF, y >= volume.dims.y))
                    return;
#else
                    if (Emulation::All(x >= volume.dims.x, cta_buffer) || Emulation::All(y >= volume.dims.y, cta_buffer))
                        return;
#endif

                    float3 V;
                    V.x = (x + 0.5f) * volume.voxel_size.x;
                    V.y = (y + 0.5f) * volume.voxel_size.y;

                    int ftid = Block::flattenedThreadId();

                    for (int z = 0; z < volume.dims.z - 1; ++z)
                    {
                        float3 points[MAX_LOCAL_POINTS];
                        int local_count = 0;

                        if (x < volume.dims.x && y < volume.dims.y)
                        {
                            int W;
                            float F = fetch(x, y, z, W);

                            if (W != 0 && F != 1.f)
                            {
                                V.z = (z + 0.5f) * volume.voxel_size.z;

                                //process dx
                                if (x + 1 < volume.dims.x)
                                {
                                    int Wn;
                                    float Fn = fetch(x + 1, y, z, Wn);

                                    if (Wn != 0 && Fn != 1.f)
                                        if ((F > 0 && Fn < 0) || (F < 0 && Fn > 0))
                                        {
                                            float3 p;
                                            p.y = V.y;
                                            p.z = V.z;

                                            float Vnx = V.x + volume.voxel_size.x;

                                            float d_inv = 1.f / (fabs(F) + fabs(Fn));
                                            p.x = (V.x * fabs(Fn) + Vnx * fabs(F)) * d_inv;

                                            points[local_count++] = aff * p;
                                        }
                                } /* if (x + 1 < volume.dims.x) */

                                //process dy
                                if (y + 1 < volume.dims.y)
                                {
                                    int Wn;
                                    float Fn = fetch(x, y + 1, z, Wn);

                                    if (Wn != 0 && Fn != 1.f)
                                        if ((F > 0 && Fn < 0) || (F < 0 && Fn > 0))
                                        {
                                            float3 p;
                                            p.x = V.x;
                                            p.z = V.z;

                                            float Vny = V.y + volume.voxel_size.y;

                                            float d_inv = 1.f / (fabs(F) + fabs(Fn));
                                            p.y = (V.y * fabs(Fn) + Vny * fabs(F)) * d_inv;

                                            points[local_count++] = aff * p;
                                        }
                                } /*  if (y + 1 < volume.dims.y) */

                                //process dz
                                //if (z + 1 < volume.dims.z) // guaranteed by loop
                                {
                                    int Wn;
                                    float Fn = fetch(x, y, z + 1, Wn);

                                    if (Wn != 0 && Fn != 1.f)
                                        if ((F > 0 && Fn < 0) || (F < 0 && Fn > 0))
                                        {
                                            float3 p;
                                            p.x = V.x;
                                            p.y = V.y;

                                            float Vnz = V.z + volume.voxel_size.z;

                                            float d_inv = 1.f / (fabs(F) + fabs(Fn));
                                            p.z = (V.z * fabs(Fn) + Vnz * fabs(F)) * d_inv;

                                            points[local_count++] = aff * p;
                                        }
                                } /* if (z + 1 < volume.dims.z) */
                            } /* if (W != 0 && F != 1.f) */
                        } /* if (x < volume.dims.x && y < volume.dims.y) */

#if __CUDA_ARCH__ >= 200
                        ///not we fulfilled points array at current iteration
                        int total_warp = __popc (__ballot_sync(0xFFFFFFFF, local_count > 0)) + __popc (__ballot_sync(0xFFFFFFFF, local_count > 1)) + __popc (__ballot_sync(0xFFFFFFFF, local_count > 2));
#else
                        int tid = Block::flattenedThreadId();
                        cta_buffer[tid] = local_count;
                        int total_warp = Emulation::warp_reduce(cta_buffer, tid);
#endif
                        __shared__ float storage_X[CTA_SIZE * MAX_LOCAL_POINTS];
                        __shared__ float storage_Y[CTA_SIZE * MAX_LOCAL_POINTS];
                        __shared__ float storage_Z[CTA_SIZE * MAX_LOCAL_POINTS];

                        if (total_warp > 0)
                        {
                            int lane = Warp::laneId();
                            int storage_index = (ftid >> Warp::LOG_WARP_SIZE) * Warp::WARP_SIZE * MAX_LOCAL_POINTS;

                            volatile int* cta_buffer = (int*) (storage_X + storage_index);

                            cta_buffer[lane] = local_count;
                            int offset = scan_warp<exclusive>(cta_buffer, lane);

                            if (lane == 0)
                            {
                                int old_global_count = atomicAdd(&global_count, total_warp);
                                cta_buffer[0] = old_global_count;
                            }
                            int old_global_count = cta_buffer[0];

                            for (int l = 0; l < local_count; ++l)
                            {
                                storage_X[storage_index + offset + l] = points[l].x;
                                storage_Y[storage_index + offset + l] = points[l].y;
                                storage_Z[storage_index + offset + l] = points[l].z;
                            }

                            Point *pos = output.data + old_global_count + lane;
                            for (int idx = lane; idx < total_warp; idx += Warp::STRIDE, pos += Warp::STRIDE)
                            {
                                float x = storage_X[storage_index + idx];
                                float y = storage_Y[storage_index + idx];
                                float z = storage_Z[storage_index + idx];
                                *pos = make_float4(x, y, z, 0.f);
                            }

                            bool full = (old_global_count + total_warp) >= output.size;

                            if (full)
                                break;
                        }

                    } /* for(int z = 0; z < volume.dims.z - 1; ++z) */

                    ///////////////////////////
                    // prepare for future scans
                    if (ftid == 0)
                    {
                        unsigned int total_blocks = gridDim.x * gridDim.y * gridDim.z;
                        unsigned int value = atomicInc(&blocks_done, total_blocks);

                        //last block
                        if (value == total_blocks - 1)
                        {
                            output_count = min((int) output.size, global_count);
                            blocks_done = 0;
                            global_count = 0;
                        }
                    }
                }
        };

        __global__ void extract_kernel(const FullScan6 fs, PtrSz<Point> output)
        {
            fs(output);
        }

        struct ExtractNormals
        {
                typedef float8 float8;

                TsdfVolume volume;
                PtrSz<Point> points;
                float3 voxel_size_inv;
                float3 gradient_delta;
                Aff3f aff;
                Mat3f Rinv;

                ExtractNormals(const TsdfVolume& vol) :
                        volume(vol)
                {
                    voxel_size_inv.x = 1.f / volume.voxel_size.x;
                    voxel_size_inv.y = 1.f / volume.voxel_size.y;
                    voxel_size_inv.z = 1.f / volume.voxel_size.z;
                }

                __kf_device__
                int3 getVoxel(const float3& p) const
                {
                    //rounding to nearest even
                    int x = __float2int_rn(p.x * voxel_size_inv.x);
                    int y = __float2int_rn(p.y * voxel_size_inv.y);
                    int z = __float2int_rn(p.z * voxel_size_inv.z);
                    return make_int3(x, y, z);
                }

                __kf_device__
                void operator ()(float4* output) const
                {
                    int idx = threadIdx.x + blockIdx.x * blockDim.x;

                    if (idx >= points.size)
                        return;

                    const float qnan = numeric_limits<float>::quiet_NaN();
                    float3 n = make_float3(qnan, qnan, qnan);

                    float3 point = Rinv * (tr(points.data[idx]) - aff.t);
                    int3 g = getVoxel(point);

                    if (g.x > 1 && g.y > 1 && g.z > 1 && g.x < volume.dims.x - 2 && g.y < volume.dims.y - 2 && g.z < volume.dims.z - 2)
                    {
                        float3 t;

                        t = point;
                        t.x += gradient_delta.x;
                        ;
                        float Fx1 = interpolate(volume, t * voxel_size_inv);

                        t = point;
                        t.x -= gradient_delta.x;
                        float Fx2 = interpolate(volume, t * voxel_size_inv);

                        n.x = __fdividef(Fx1 - Fx2, gradient_delta.x);

                        t = point;
                        t.y += gradient_delta.y;
                        float Fy1 = interpolate(volume, t * voxel_size_inv);

                        t = point;
                        t.y -= gradient_delta.y;
                        float Fy2 = interpolate(volume, t * voxel_size_inv);

                        n.y = __fdividef(Fy1 - Fy2, gradient_delta.y);

                        t = point;
                        t.z += gradient_delta.z;
                        float Fz1 = interpolate(volume, t * voxel_size_inv);

                        t = point;
                        t.z -= gradient_delta.z;
                        float Fz2 = interpolate(volume, t * voxel_size_inv);

                        n.z = __fdividef(Fz1 - Fz2, gradient_delta.z);

                        n = normalized(aff.R * n);
                    }

                    output[idx] = make_float4(n.x, n.y, n.z, 0);
                }
        };

        __global__ void extract_normals_kernel(const ExtractNormals en, float4* output)
        {
            en(output);
        }
    }
}

size_t kfusion::device::extractCloud(const TsdfVolume& volume, const Aff3f& aff, PtrSz<Point> output)
{
    typedef FullScan6 FS;
    FS fs(volume);
    fs.aff = aff;

    dim3 block(FS::CTA_SIZE_X, FS::CTA_SIZE_Y);
    dim3 grid(divUp(volume.dims.x, block.x), divUp(volume.dims.y, block.y));

    extract_kernel<<<grid, block>>>(fs, output);
    cudaSafeCall(hipGetLastError());
    cudaSafeCall(hipDeviceSynchronize());

    int size;
    cudaSafeCall(hipMemcpyFromSymbol(&size, HIP_SYMBOL(output_count), sizeof(size)));
    return (size_t) size;
}

void kfusion::device::extractNormals(const TsdfVolume& volume, const PtrSz<Point>& points, const Aff3f& aff, const Mat3f& Rinv, float gradient_delta_factor, float4* output)
{
    ExtractNormals en(volume);
    en.points = points;
    en.gradient_delta = volume.voxel_size * gradient_delta_factor;
    en.aff = aff;
    en.Rinv = Rinv;

    dim3 block(256);
    dim3 grid(divUp((int) points.size, block.x));

    extract_normals_kernel<<<grid, block>>>(en, output);
    cudaSafeCall(hipGetLastError());
    cudaSafeCall(hipDeviceSynchronize());
}

namespace kfusion
{
	namespace device
	{
		__kf_device__ int compute_index(TsdfVolume& volume, int x, int y, int z, int min_weight, float values[8])
		{
			int weight;
			int index = 0;
			index =  int((values[0] = unpack_tsdf(*volume(x + 0, y + 0, z + 0), weight)) < 0.0);
			if (weight < min_weight) return 0;
			index += int((values[1] = unpack_tsdf(*volume(x + 1, y + 0, z + 0), weight)) < 0.0) * 2;
			if (weight < min_weight) return 0;
			index += int((values[2] = unpack_tsdf(*volume(x + 1, y + 1, z + 0), weight)) < 0.0) * 4;
			if (weight < min_weight) return 0;
			index += int((values[3] = unpack_tsdf(*volume(x + 0, y + 1, z + 0), weight)) < 0.0) * 8;
			if (weight < min_weight) return 0;
			index += int((values[4] = unpack_tsdf(*volume(x + 0, y + 0, z + 1), weight)) < 0.0) * 16;
			if (weight < min_weight) return 0;
			index += int((values[5] = unpack_tsdf(*volume(x + 1, y + 0, z + 1), weight)) < 0.0) * 32;
			if (weight < min_weight) return 0;
			index += int((values[6] = unpack_tsdf(*volume(x + 1, y + 1, z + 1), weight)) < 0.0) * 64;
			if (weight < min_weight) return 0;
			index += int((values[7] = unpack_tsdf(*volume(x + 0, y + 1, z + 1), weight)) < 0.0) * 128;
			if (weight < min_weight) return 0;
			return index;
		}
		__global__ void classify_voxels_kernel(TsdfVolume volume, int min_weight, PtrSz<int> numVertsTable,
			                                   PtrSz<uchar> voxelVertices, PtrSz<uchar> voxelOccupied)
		{
			int x = blockIdx.x * blockDim.x + threadIdx.x;
			int y = blockIdx.y * blockDim.y + threadIdx.y;

			if (x >= volume.dims.x-1 || y >= volume.dims.y-1)
				return;

			for (int z = 0; z < volume.dims.z-1; ++z) {
				float values[8];
				int index = compute_index(volume, x, y, z, min_weight, values);
				int i = x + y * (volume.dims.x-1) + z * (volume.dims.x-1) * (volume.dims.y-1);
				int nVerts = numVertsTable[index];
				voxelVertices[i] = nVerts;
				voxelOccupied[i] = nVerts > 0;
			}
		}

		__global__ void compact_voxels_kernel(PtrSz<uchar> voxelOccupied, PtrSz<unsigned int> voxelOccupiedScan,
			                                  int3 dims, PtrSz<unsigned int> voxelOccupiedCompact)
		{
			int x = blockIdx.x * blockDim.x + threadIdx.x;
			int y = blockIdx.y * blockDim.y + threadIdx.y;

			if (x >= dims.x - 1 || y >= dims.y - 1)
				return;

			for (int z = 0; z < dims.z - 1; ++z) {
				int i = x + y * (dims.x - 1) + z * (dims.x - 1) * (dims.y - 1);
				if (voxelOccupied[i]) {
					voxelOccupiedCompact[voxelOccupiedScan[i]] = i;
				}
			}
		}
		__kf_device__
		float3 vertex_interpolate(const float3 p0, const float3 p1, const float f0, const float f1)
		{
			float t = (0.f - f0) / (f1 - f0 + 1e-15f);
			return make_float3(p0.x + t * (p1.x - p0.x),
				               p0.y + t * (p1.y - p0.y),
			                   p0.z + t * (p1.z - p0.z));
		}


		__global__ void generate_triangles_kernel(TsdfVolume volume, int min_weight,
			                                      PtrSz<unsigned int> voxelOccupiedCompact,
			                                      PtrSz<unsigned int> voxelVerticesScan,
			                                      PtrSz<int> numVertsTable, PtrSz<int> triangleTable,
			                                      PtrSz<float3> output)
		{
			int i = (blockIdx.y * 65536 + blockIdx.x) * 256 + threadIdx.x;
			if (i >= voxelOccupiedCompact.size) {
				i = voxelOccupiedCompact.size - 1;
			}

			int voxel = voxelOccupiedCompact[i];
			const int z = voxel / ((volume.dims.x-1) * (volume.dims.y-1));
			const int y = (voxel - z * (volume.dims.x-1) * (volume.dims.y-1)) / (volume.dims.x-1);
			const int x = (voxel - z * (volume.dims.x-1) * (volume.dims.y-1)) - y * (volume.dims.x-1);

			float3 p = make_float3(x, y, z) * volume.voxel_size;

			float3 v[8];
			v[0] = p;
			v[1] = p + make_float3(volume.voxel_size.x, 0, 0);
			v[2] = p + make_float3(volume.voxel_size.x, volume.voxel_size.y, 0);
			v[3] = p + make_float3(0, volume.voxel_size.y, 0);
			v[4] = p + make_float3(0, 0, volume.voxel_size.z);
			v[5] = p + make_float3(volume.voxel_size.x, 0, volume.voxel_size.z);
			v[6] = p + make_float3(volume.voxel_size.x, volume.voxel_size.y, volume.voxel_size.z);
			v[7] = p + make_float3(0, volume.voxel_size.y, volume.voxel_size.z);

			float values[8];
			int index = compute_index(volume, x, y, z, min_weight, values);

			__shared__ float3 vertex_list[12][256];
			vertex_list[0][threadIdx.x] = vertex_interpolate(v[0], v[1], values[0], values[1]);
			vertex_list[1][threadIdx.x] = vertex_interpolate(v[1], v[2], values[1], values[2]);
			vertex_list[2][threadIdx.x] = vertex_interpolate(v[2], v[3], values[2], values[3]);
			vertex_list[3][threadIdx.x] = vertex_interpolate(v[3], v[0], values[3], values[0]);
			vertex_list[4][threadIdx.x] = vertex_interpolate(v[4], v[5], values[4], values[5]);
			vertex_list[5][threadIdx.x] = vertex_interpolate(v[5], v[6], values[5], values[6]);
			vertex_list[6][threadIdx.x] = vertex_interpolate(v[6], v[7], values[6], values[7]);
			vertex_list[7][threadIdx.x] = vertex_interpolate(v[7], v[4], values[7], values[4]);
			vertex_list[8][threadIdx.x] = vertex_interpolate(v[0], v[4], values[0], values[4]);
			vertex_list[9][threadIdx.x] = vertex_interpolate(v[1], v[5], values[1], values[5]);
			vertex_list[10][threadIdx.x] = vertex_interpolate(v[2], v[6], values[2], values[6]);
			vertex_list[11][threadIdx.x] = vertex_interpolate(v[3], v[7], values[3], values[7]);
			__syncthreads();

			int nVerts = numVertsTable[index];
			for (int v = 0; v < nVerts; v += 3) {
				const int offset = voxelVerticesScan[voxel] + v;

				const int v1 = triangleTable[(index * 16) + v + 0];
				const int v2 = triangleTable[(index * 16) + v + 1];
				const int v3 = triangleTable[(index * 16) + v + 2];

				output[offset + 0] = vertex_list[v1][threadIdx.x];
				output[offset + 1] = vertex_list[v2][threadIdx.x];
				output[offset + 2] = vertex_list[v3][threadIdx.x];
			}

		}

	}
}

unsigned int do_exclusive_scan(DeviceArray<uchar>& input, DeviceArray<unsigned int>& output)
{
	thrust::exclusive_scan(thrust::device_ptr<uchar>(input.ptr()),
                           thrust::device_ptr<uchar>(input.ptr() + input.size()),
                           thrust::device_ptr<unsigned int>(output.ptr()));

	uchar lastElement;
	cudaSafeCall(hipMemcpy(&lastElement, input.ptr() + input.size() - 1,
		                    sizeof(uchar), hipMemcpyDeviceToHost));
	unsigned int lastElementScan;
	cudaSafeCall(hipMemcpy(&lastElementScan, output.ptr() + output.size() - 1,
	                        sizeof(unsigned int), hipMemcpyDeviceToHost));

	return lastElement + lastElementScan;
}

DeviceArray<float3> kfusion::device::marchingCubes(const TsdfVolume& volume, int min_weight)
{
	// Constant table with number of vertices per cube configuration
	DeviceArray<int> numVertsTable(sizeof(yak::numVertsTable));
	numVertsTable.upload(yak::numVertsTable, sizeof(yak::numVertsTable)/sizeof(yak::numVertsTable[0]));

	// Identify all occupied voxels and number of vertices
	// produced by each occupied voxel
	int numVoxels = (volume.dims.x-1) * (volume.dims.y-1) * (volume.dims.z-1);
	DeviceArray<uchar> voxelVertices(numVoxels);
	DeviceArray<uchar> voxelOccupied(numVoxels);

	dim3 block(32, 8);
	dim3 grid(1, 1, 1);
	grid.x = divUp(volume.dims.x, block.x);
	grid.y = divUp(volume.dims.y, block.y);

	classify_voxels_kernel<<<grid, block>>>(volume, min_weight, numVertsTable, voxelVertices, voxelOccupied);
	cudaSafeCall(hipGetLastError());

	// Perform exclusive scan of the occupied voxels information in order to
	// determine total number of occupied voxels
	DeviceArray<unsigned int> voxelOccupiedScan(numVoxels);
	unsigned int activeVoxels = do_exclusive_scan(voxelOccupied, voxelOccupiedScan);

	// Create an array with only the voxels that are actually occupied
	DeviceArray<unsigned int> voxelOccupiedCompact(activeVoxels);
	compact_voxels_kernel<<<grid, block>>>(voxelOccupied, voxelOccupiedScan, volume.dims, voxelOccupiedCompact);

	// Perform an exclusive scan of the array with number of vertices produced
	// by each voxel. These values are used for indexing of the final output.
	DeviceArray<unsigned int> voxelVerticesScan(numVoxels);
	unsigned int numVertices = do_exclusive_scan(voxelVertices, voxelVerticesScan);

	// Table with edges composing the triangle for each cube configuration
	DeviceArray<int> triangleTable(sizeof(yak::triangleTable));
	triangleTable.upload(&yak::triangleTable[0][0], sizeof(yak::triangleTable) / sizeof(int));

	// Produce triangles from each occupied voxel
	DeviceArray<float3> output(numVertices);

	const int n_threads = 256;
	dim3 genBlock(n_threads);
	unsigned blocks_num = divUp(activeVoxels, n_threads);
	dim3 genGrid(min(blocks_num, 65536), divUp(blocks_num, 65536));

	generate_triangles_kernel<<<genGrid, genBlock>>>(volume, min_weight, voxelOccupiedCompact, voxelVerticesScan,
		                                             numVertsTable, triangleTable, output);
	cudaSafeCall(hipGetLastError());


	return output;
}
